#include "hip/hip_runtime.h"
#include "ArgbColor.cuh"

//__device__ ArgbColor ArgbColor::White = ArgbColor(0xFF, 0xFF, 0xFF, 0xFF);
//__constant__ ArgbColor ArgbColor::Black = ArgbColor(0xFF, 0x00, 0x00, 0x00);
//__constant__ ArgbColor ArgbColor::Transparent = ArgbColor(0x00, 0xFF, 0xFF, 0xFF);

__device__ ArgbColor ArgbColor::fromArgb(uchar4 x) {
  ArgbColor c;
  c.b = (uint8_t) x.x;
  c.g = (uint8_t) x.y;
  c.r = (uint8_t) x.z;
  c.a = 0xFF;

  return c;
}