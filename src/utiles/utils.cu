#include "hip/hip_runtime.h"
#include "utils.cuh"

__host__ __device__ void ToCartesian(const float r, const float theta, const float phi,
									 float &x, float &y, float &z) {
  x = r * cos(phi) * sin(theta);
  y = r * sin(phi) * sin(theta);
  z = r * cos(theta);
}

__host__ __device__ void ToSpherical(const float x, const float y, const float z,
									 float &r, float &theta, float &phi) {
  r = sqrt(x * x + y * y + z * z);
  phi = atan2(y, x);
  theta = acos(z / r);
}

__host__ __device__ void ToSpherical(const float3 v, float &r,
									 float &theta, float &phi) {
  r = sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
  phi = atan2(v.y, v.x);
  theta = acos(v.z / r);
}

__host__ __device__ float DoubleMod(float n, float m) {
  float x = floor(n / m);
  return n - (m * x);
}

__host__ __device__ float GetBrightness(ArgbColor c) {
  float r = (float)c.r / 255.0f;
  float g = (float)c.g / 255.0f;
  float b = (float)c.b / 255.0f;

  float max, min;

  max = r;
  min = r;

  if (g > max) max = g;
  if (b > max) max = b;

  if (g < min) min = g;
  if (b < min) min = b;

  return (max + min) / 2.0;
}

__host__ __device__ int Cap(int x, int max) {
  if (x > max) {
	return max;
  } else {
	return x;
  }
}

__host__ __device__ int CapMin(int x, int min) {
  if (x < min) {
	return min;
  } else {
	return x;
  }
}

__host__ __device__ ArgbColor AddColor(ArgbColor hitColor, ArgbColor tintColor) {
  if (tintColor == ArgbColor(0x00, 0xFF, 0xFF, 0xFF)) {
	return hitColor;
  }
  float brightness = GetBrightness(tintColor);
  ArgbColor c;
  c.r = (uint8_t)Cap((int)(((1.0 - brightness) * hitColor.r) +
	  CapMin(tintColor.r, 0) * 255 / 205
  ), 255);
  c.g = (uint8_t)Cap((int)(((1.0 - brightness) * hitColor.g) +
	  CapMin(tintColor.g, 0) * 255 / 205
  ), 255);
  c.b = (uint8_t)Cap((int)(((1.0 - brightness) * hitColor.b) +
	  CapMin(tintColor.b, 0) * 255 / 205
  ), 255);
  c.a = (uint8_t)0xFF;
  return c;
}

__host__ __device__ void SphericalMap(int SizeX,
									  int SizeY,
									  const float r,
									  const float theta,
									  const float phi,
									  int &x,
									  int &y) {
  x = (int)((phi / (2.0 * M_PI)) * SizeX) % SizeX;
  y = (int)((theta / M_PI) * SizeY) % SizeY;
  if (x < 0) { x = SizeX + x; }
  if (y < 0) { y = SizeY + y; }
}

__host__ __device__ void DiskMap(float rMin,
								 float rMax,
								 int SizeX,
								 int SizeY,
								 const float r,
								 const float theta,
								 const float phi,
								 int &x,
								 int &y) {
  if (r < rMin || r > rMax) {
	x = 0;
	y = SizeY;
  }

  x = (int)((phi / (2 * M_PI)) * SizeX) % SizeX;
  if (x < 0) { x = SizeX + x; }
  y = (int)(((r - rMin) / (rMax - rMin)) * SizeY);
  if (y > SizeY - 1) { y = SizeY - 1; }
}

__host__ cv::Mat getNativeTextureBitmap(cv::Mat texture) {
  return texture.clone();
}
