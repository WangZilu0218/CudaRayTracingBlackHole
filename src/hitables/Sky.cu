#include "hip/hip_runtime.h"
//
// Created by 王子路 on 2022/5/13.
//

#include <iostream>
#include "Sky.cuh"
#include "src/utiles/utils.cuh"
#include "hip/hip_runtime_api.h"
using namespace std;
using namespace cv;

__host__ Sky::Sky(Mat texture, float radius) {
  this->radius = radius;
  radiusSqr = radius * radius;
  if (!texture.empty()) {
	textureWidth = texture.cols;
	textureHeight = texture.rows;
	textureBitmap = getNativeTextureBitmap(texture);
  }
}

__host__ void Sky::SetAttributes(int gpuId) {
  hipSetDevice(gpuId);
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
//  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();

  const int spitch = textureBitmap.channels() * textureWidth * sizeof(unsigned char);
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;

  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModePoint;
//  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;


  checkCudaErrors(hipMallocArray(&array_texture, &channelDesc, textureWidth, textureHeight));

  checkCudaErrors(hipMemcpy2DToArray(array_texture,
									  0,
									  0,
									  textureBitmap.ptr(),
									  spitch,
									  textureBitmap.channels() * textureWidth * sizeof(unsigned char),
									  textureHeight,
									  hipMemcpyHostToDevice));
  resDesc.res.array.array = array_texture;

  checkCudaErrors(hipCreateTextureObject(&text_obj, &resDesc, &texDesc, NULL));
}

__host__ void Sky::ReleaseAttributes(int gpuId) {
  hipSetDevice(gpuId);
  checkCudaErrors(hipFreeArray(array_texture));
  checkCudaErrors(hipDestroyTextureObject(text_obj));
}

__device__ ArgbColor Sky::GetColor(int side, float r, float theta, float phi, int x, int y) {
  return ArgbColor(0xFF, 0xFF, 0xFF, 0xFF);
}

__host__ __device__ Sky *Sky::SetTextureOffset(float offset) {
  textureOffset = offset;
  return this;
}

__device__ bool Sky::Hit(float3 &point, float sqrNorm, float3 &prevPoint, float prevSqrNorm,
						 float3 &velocity, SchwarzschildBlackHoleEquation *equation, float r, float theta,
						 float phi, ArgbColor &color, bool &stop, bool debug, int x, int y) {
  // Has the ray escaped to infinity?
  if (sqrNorm > radiusSqr) {
	int xPos, yPos;
	SphericalMap(textureWidth, textureHeight, r, theta, phi, xPos, yPos);
	color = AddColor(ArgbColor::fromArgb(tex2D<uchar4>(text_obj, xPos, yPos)), color);
	stop = true;
	return true;
  }
  return false;
}

//Vector3D Sky::IntersectionSearch(Vector3D prevPoint, Vector3D velocity, SchwarzschildBlackHoleEquation *equation) {
//    float stepLow = 0., stepHigh = equation.StepSize;
//    Vector3D newPoint = prevPoint;
//    Vector3D tempVelocity;
//    while (true) {
//        float stepMid = (stepLow + stepHigh) / 2.;
//        newPoint = prevPoint;
//        tempVelocity = velocity;
//        equation->Function(newPoint, tempVelocity, stepMid);
//
//        double distance = newPoint.norm2();
//        if (abs(stepHigh - stepLow) < 0.00001) {
//            break;
//        }
//        if (distance > radius) {
//            stepHigh = stepMid;
//        }
//        else {
//            stepLow = stepMid;
//        }
//    }
//    return newPoint;
//}