#include "hip/hip_runtime.h"
//
// Created by 王子路 on 2022/5/13.
//

#include "TexturedDisk.cuh"
#include "src/utiles/utils.cuh"
#include "hip/hip_runtime_api.h"

__host__ TexturedDisk::TexturedDisk(float radiusInner, float radiusOuter, Mat texture) {
  this->radiusInner = radiusInner;
  this->radiusOuter = radiusOuter;
  radiusInnerSqr = radiusInner * radiusInner;
  radiusOuterSqr = radiusOuter * radiusOuter;
  textureWidth = texture.cols;
  textureHeight = texture.rows;
  textureBitmap = texture.clone();
}

__host__ void TexturedDisk::SetAttributes(int gpuId) {
  hipSetDevice(gpuId);
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
  const int spitch = textureBitmap.channels() * textureWidth * sizeof(unsigned char);
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;

  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModePoint;
//  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  checkCudaErrors(hipMallocArray(&array_texture_disk, &channelDesc, textureWidth, textureHeight));
  checkCudaErrors(hipMemcpy2DToArray(array_texture_disk,
									  0,
									  0,
									  textureBitmap.ptr(),
									  spitch,
									  textureBitmap.channels() * textureWidth * sizeof(unsigned char),
									  textureHeight,
									  hipMemcpyHostToDevice));
  resDesc.res.array.array = array_texture_disk;
  checkCudaErrors(hipCreateTextureObject(&tex_obj, &resDesc, &texDesc, NULL));
}

__host__ void TexturedDisk::ReleaseAtrributes(int gpuId) {
  hipSetDevice(gpuId);
  checkCudaErrors(hipFreeArray(array_texture_disk));
  checkCudaErrors(hipDestroyTextureObject(tex_obj));
}


__device__ ArgbColor TexturedDisk::GetColor(int side, float r, float theta, float phi, int x, int y) {
  int xPos, yPos;
  DiskMap(radiusInner, radiusOuter, textureWidth, textureHeight, r, theta, phi, xPos, yPos);
  return ArgbColor::fromArgb(tex2D<uchar4>(tex_obj, xPos, yPos));
  // row major order so like this apparently. needs testing
  // original code: return Color.FromArgb(textureBitmap[yPos * textureWidth + xPos]);
}

__device__ bool TexturedDisk::Hit(float3 &point, float sqrNorm, float3 &prevPoint, float prevSqrNorm,
								  float3 &velocity, SchwarzschildBlackHoleEquation *equation, float r, float theta,
								  float phi, ArgbColor &color, bool &stop, bool debug, int x, int y) {
  // Remember what side of the plane we're currently on, so that we can detect
  // whether we've crossed the plane after stepping.
  int side = prevPoint.y > 0 ? -1 : prevPoint.y < 0 ? 1 : 0;

  // Did we cross the horizontal plane?
  bool success = false;
  if (point.y * side >= 0) {
	float3 colpoint = IntersectionSearch(side, prevPoint, velocity, equation);
	float colpointsqr = dot(colpoint, colpoint);
	if ((colpointsqr >= radiusInnerSqr) && (colpointsqr <= radiusOuterSqr)) {
	  float tempR = 0;
	  float tempTheta = 0;
	  float tempPhi = 0;
	  ToSpherical(colpoint, tempR, tempTheta, tempPhi);
	  color = AddColor(GetColor(side, tempR, tempPhi, tempTheta + M_PI / 12, x, y), color);
	  stop = false;
	  success = true;
	}
  }
  return success;
}

__host__ __device__ float3 TexturedDisk::IntersectionSearch(int side,
															float3 prevPoint,
															float3 velocity,
															SchwarzschildBlackHoleEquation *equation) {
  float stepLow = 0, stepHigh = equation->StepSize;
  float3 newPoint = prevPoint;
  float3 tempVelocity;
  while (true) {
	float stepMid = (stepLow + stepHigh) / 2;
	newPoint = prevPoint;
	tempVelocity = velocity;
	equation->Function(newPoint, tempVelocity, stepMid);
	if (abs(stepHigh - stepLow) < 0.00001) {
	  break;
	}
	if (side * newPoint.y > 0) {
	  stepHigh = stepMid;
	} else {
	  stepLow = stepMid;
	}
  }
  return newPoint;
}
