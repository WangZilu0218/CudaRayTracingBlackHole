#include "hip/hip_runtime.h"
//
// Created by 王子路 on 2022/5/13.
//
#include "Horizon.cuh"
#include "src/utiles/utils.cuh"
#include "hip/hip_runtime_api.h"

using namespace std;
using namespace cv;

__host__ Horizon::Horizon(Mat texture, bool checkered) {
  this->checkered = checkered;
  if (!texture.empty()) {
	cout << "horizon texture is not empty" << endl;
	textureWidth = texture.cols;
	textureHeight = texture.rows;
	textureBitmap = texture.clone();
	textureBitmapIsNull = false;
  }
}

__host__ void Horizon::SetAttributes(int gpuId) {
  hipSetDevice(gpuId);
  if (!textureBitmapIsNull) {
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
	const int spitch = textureBitmap.channels() * textureWidth * sizeof(unsigned char);
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;

	printf("horizon texture width:%d\n", textureWidth);
	printf("horizon texture height:%d\n", textureHeight);

	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModePoint;
//  texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;
	hipMallocArray(&array_texture, &channelDesc, textureWidth, textureHeight);
	hipMemcpy2DToArray(array_texture,
						0,
						0,
						textureBitmap.ptr(0),
						spitch,
						textureBitmap.channels() * textureWidth * sizeof(unsigned char),
						textureHeight,
						hipMemcpyHostToDevice);
	resDesc.res.array.array = array_texture;
	hipCreateTextureObject(&tex_obj, &resDesc, &texDesc, NULL);
  }

};

__device__ ArgbColor Horizon::GetColor(int side, float r, float theta, float phi, int x, int y) {
  return ArgbColor(0xFF, 0xFF, 0xFF, 0xFF);
}

__host__ void Horizon::ReleaseAttributes(int gpuId) {
  hipSetDevice(gpuId);
  if (!textureBitmapIsNull) {
	checkCudaErrors(hipFreeArray(array_texture));
	checkCudaErrors(hipDestroyTextureObject(tex_obj));
  }
}

__device__ bool Horizon::Hit(float3 &point, float sqrNorm, float3 &prevPoint, float prevSqrNorm,
							 float3 &velocity, SchwarzschildBlackHoleEquation *equation, float r, float theta,
							 float phi, ArgbColor &color, bool &stop, bool debug, int x, int y) {

  // Has the ray fallen past the horizon?
  if (prevSqrNorm > 1 && sqrNorm < 1) {
	float3 colpoint = IntersectionSearch(prevPoint, velocity, equation);

	float tempR = 0., tempTheta = 0., tempPhi = 0.;
	ToSpherical(colpoint, tempR, tempTheta, tempPhi);

	ArgbColor col = ArgbColor(0xFF, 0x00, 0x00, 0x00);
	if (checkered) {
	  float m1 = DoubleMod(tempTheta, 1.04719); // Pi / 3
	  float m2 = DoubleMod(tempPhi, 1.04719); // Pi / 3
	  if ((m1 < 0.52359) ^ (m2 < 0.52359)) { // Pi / 6
		//col = Color.Green
		col = ArgbColor(0xFF, 0x00, 0x80, 0x00);
	  }
	} else if (!textureBitmapIsNull) {
//	  cout << "texturebitmap for horizon not null" << endl;
	  int xPos, yPos;
	  SphericalMap(textureWidth, textureHeight, r, theta, -phi, xPos, yPos);
	  col = ArgbColor::fromArgb(make_uchar4(0, 0, 0, 0));
	}
	color = AddColor(col, color);
	//cout << color << endl;
	stop = true;
	return true;
  }
  return false;
}

__host__ __device__ float3 Horizon::IntersectionSearch(float3 prevPoint, float3 velocity,
													   SchwarzschildBlackHoleEquation *equation) {
  float stepLow = 0, stepHigh = equation->StepSize;
  float3 newPoint = prevPoint;
  float3 tempVelocity;
  while (true) {
	float stepMid = (stepLow + stepHigh) / 2;
	newPoint = prevPoint;
	tempVelocity = velocity;
	equation->Function(newPoint, tempVelocity, stepMid);

	double distance = dot(newPoint, newPoint);
	if (abs(stepHigh - stepLow) < 0.00001) {
	  break;
	}
	if (distance < radius) {
	  stepHigh = stepMid;
	} else {
	  stepLow = stepMid;
	}
  }
  return newPoint;
}
