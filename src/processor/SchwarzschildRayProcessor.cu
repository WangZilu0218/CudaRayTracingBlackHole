#include "hip/hip_runtime.h"
//
// Created by 王子路 on 2022/5/13.
//

#include "hip/hip_vector_types.h"
#include "src/hitables/Sky.cuh"
#include "src/hitables/Horizon.cuh"
#include "src/hitables/TexturedDisk.cuh"
#include "src/utiles/ArgbColor.cuh"
#include "src/utiles/GPUParams.h"
#include "src/utiles/utils.cuh"
#include "hip/hip_vector_types.h"
#include "SchwarzschildRayProcessor.cuh"
#include "hip/hip_cooperative_groups.h"

using namespace cooperative_groups;
namespace cg = cooperative_groups;

#define NUMBERITERATION 10000

__global__ void renderBlackHoleKernel(uchar3 *bitmap,
									  unsigned int linesThisLoop,
									  unsigned int width,
									  unsigned int height,
									  float tanFov,
									  float4 *matrixData,
									  float3 CameraPosition,
									  Sky sky,
									  TexturedDisk texturedDisk,
									  Horizon horizon,
									  SchwarzschildBlackHoleEquation equation,
									  int gpu_id,
									  unsigned int offset) {
  cg::grid_group g = cg::this_grid();
  cg::thread_block b = cg::this_thread_block();
  dim3 grid_index = b.group_index();
  dim3 block_dim = b.group_dim();
  dim3 thread_index = b.thread_index();

  unsigned int gtx = thread_index.x + block_dim.x * grid_index.x;
  unsigned int gty = thread_index.y + block_dim.y * grid_index.y;

  while (gtx < linesThisLoop) {
	gty = thread_index.y + block_dim.y * grid_index.y;
	while (gty < width) {
	  bool debug = false;
	  ArgbColor color;
	  float3 point, prevPoint;
	  float sqrNorm, prevSqrNorm;
	  float tempR = 0, tempTheta = 0, tempPhi = 0;
	  bool stop = false;
	  color = ArgbColor(0x00, 0xFF, 0xFF, 0xFF);

	  float4 view = make_float4((((float)gty) / width - 0.5f) * tanFov,
								((-(float)(gtx + offset) / height + 0.5f) * height / width) * tanFov,
								(float)1.0,
								1.0);
	  view = make_float4(dot(matrixData[0], view),
						 dot(matrixData[1], view),
						 dot(matrixData[2], view),
						 dot(matrixData[3], view));

	  float3 normView = normalize(make_float3(view.x, view.y, view.z));
	  float3 velocity = make_float3(normView.x, normView.y, normView.z);
	  point = CameraPosition;
	  sqrNorm = dot(point, point);
	  stop = false;
	  equation.SetInitialConditions(point, velocity);
	  for (int iter = 0; iter < NUMBERITERATION; iter++) {
		prevPoint = point;
		prevSqrNorm = sqrNorm;
		equation.Function(point, velocity);

		sqrNorm = dot(point, point);
		ToSpherical(point, tempR, tempTheta, tempPhi);


		if (!texturedDisk.Hit(point,
							  sqrNorm,
							  prevPoint,
							  prevSqrNorm,
							  velocity,
							  &equation,
							  tempR,
							  tempTheta,
							  tempPhi,
							  color,
							  stop,
							  debug,
							  gtx,
							  gty)) {
		  if (!horizon.Hit(point,
						   sqrNorm,
						   prevPoint,
						   prevSqrNorm,
						   velocity,
						   &equation,
						   tempR,
						   tempTheta,
						   tempPhi,
						   color,
						   stop,
						   debug,
						   gtx,
						   gty)) {
			sky.Hit(point,
					sqrNorm,
					prevPoint,
					prevSqrNorm,
					velocity,
					&equation,
					tempR,
					tempTheta,
					tempPhi,
					color,
					stop,
					debug,
					gtx,
					gty);
		  }
		}
		if (stop) {
		  break;
		}
	  }

	  if (stop == false) {
		bitmap[gtx * width + gty].x = 0;
		bitmap[gtx * width + gty].y = 0;
		bitmap[gtx * width + gty].z = 0;
	  }
	  bitmap[gtx * width + gty].x = (uchar)color.b;
	  bitmap[gtx * width + gty].y = (uchar)color.g;
	  bitmap[gtx * width + gty].z = (uchar)color.r;

	  gty += g.group_dim().y * b.group_dim().y;
	}
	gtx += g.group_dim().x * b.group_dim().x;
  }
}

void renderBlackHole(uchar3 *bitmap,
					 unsigned int linesThisLoop,
					 unsigned int width,
					 unsigned int height,
					 float tanFov,
					 float4 *matrixData,
					 float3 CameraPosition,
					 Sky &sky,
					 TexturedDisk &texturedDisk,
					 Horizon &horizon,
					 SchwarzschildBlackHoleEquation equation,
					 int gpu_id,
					 unsigned int offset
) {
  dim3 _gridDim(512, 512, 1);
  dim3 _blockDim(16, 16, 1);
  renderBlackHoleKernel<<<_gridDim, _blockDim>>>(bitmap,
												 linesThisLoop,
												 width,
												 height,
												 tanFov,
												 matrixData,
												 CameraPosition,
												 sky,
												 texturedDisk,
												 horizon,
												 equation,
												 gpu_id,
												 offset);
}
